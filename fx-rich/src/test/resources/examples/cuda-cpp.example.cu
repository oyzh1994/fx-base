
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function
__global__ void addKernel(int *a, int *b, int *c, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

int main() {
  const int size = 5;

  // Host arrays
  int h_a[size] = {1, 2, 3, 4, 5};
  int h_b[size] = {5, 4, 3, 2, 1};
  int h_c[size];

  // Device arrays
  int *d_a, *d_b, *d_c;
  hipMalloc((void**)&d_a, size * sizeof(int));
  hipMalloc((void**)&d_b, size * sizeof(int));
  hipMalloc((void**)&d_c, size * sizeof(int));

  // Copy data from host to device
  hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch kernel
  addKernel<<<1, size>>>(d_a, d_b, d_c, size);

  // Copy result from device to host
  hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

  // Output result
  std::cout << "Result: ";
  for (int i = 0; i < size; ++i) {
    std::cout << h_c[i] << " ";
  }
  std::cout << std::endl;

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
